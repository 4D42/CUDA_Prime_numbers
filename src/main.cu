
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void isprime(int *test_number, int *boolprime){

  int dividedby = threadIdx.x + blockIdx.x * blockDim.x; //Compute the number wich the test_number will be divided by for each threards.

  if(dividedby > 1 && dividedby < *test_number){  // look to see if it's fine to do to test
    if(*test_number % dividedby == 0){*boolprime = 0;}
  }

}


int main(void){
  printf("Finding prime numbers using CUDA\n");

  int primelesserthan = 3000;
  int maxthreads = 1024;
  int Nb_blocks;
  int Nb_threads;

  int test_number, boolprime;  // host copies of test_number
  int *d_test_number, *d_boolprime;   // device copies of test_number

  hipMalloc((void **)&d_test_number, sizeof(test_number)); // Allocate space for device copies of test_number
  hipMalloc((void **)&d_boolprime, sizeof(boolprime));

  for(test_number = 2; test_number < primelesserthan; test_number++){

    boolprime = 1; //reset boolprime

    hipMemcpy(d_test_number, &test_number, sizeof(test_number), hipMemcpyHostToDevice);   // Copy data to device
    hipMemcpy(d_boolprime, &boolprime, sizeof(boolprime), hipMemcpyHostToDevice);

    // find the rigth number of blocks and threads

    if(test_number/maxthreads == 0){Nb_blocks = 1; Nb_threads = test_number;}
    else{Nb_blocks = test_number/maxthreads; Nb_threads = test_number%maxthreads;}


    isprime<<<Nb_blocks,Nb_threads>>>(d_test_number, d_boolprime);   // Launch add() kernel on GPU

    hipDeviceSynchronize();
    hipMemcpy(&boolprime,d_boolprime,sizeof(boolprime),hipMemcpyDeviceToHost);

    // if(boolprime == 0){printf("%d is not prime\n",test_number);}
    // else{printf("%d is prime\n",test_number);}

    if(boolprime == 1){printf("%d is prime\n",test_number);}

  }


  return 0;
}
